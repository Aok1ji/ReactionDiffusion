#include "hip/hip_runtime.h"
#include <SFML/Config.hpp>
#include <cmath>
#include <csignal>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <cudart_platform.h>

#include <iostream>

#define CUDA_CALL(x) hipError_t error = hipGetLastError(); if (error != hipSuccess) { std::cout << hipGetErrorName(error) << std::endl; std::abort(); } x
 
struct Cell {
	float a = 1, 
		  b = 0;
};

enum eChemicals {
	A, 
	B
};

static struct CudaConfigs {
	int xThreads = 80,
		yThreads = 1;
} cConfig;


static sf::Uint8 *pPixelField;
static Cell *pCurrentGrid;
static Cell *pNextGrid;
static size_t xSize; 
static size_t ySize;

static float killRate = 0.90f; 
static float feedRate = 0.80f;

static float diffusionA = 1.0f;
static float diffusionB = 0.5f;

static float dT = 1.0f;


void CudaInit(size_t width, size_t height) { 
	xSize = width;
	ySize = height;

	CUDA_CALL(hipSetDevice(0));
	hipMalloc(&pCurrentGrid, width * height * sizeof(Cell));
	hipMalloc(&pNextGrid, width * height * sizeof(Cell));
	hipMalloc(&pPixelField, width * height * sizeof(sf::Uint8) * 4);
}

void CudaExit() {
	hipFree(pCurrentGrid);
	hipFree(pNextGrid);
	hipFree(pPixelField);
}

__device__ float laplacian(eChemicals type, int x, int y, Cell* pCurrentGrid, size_t xSize, size_t ySize) {
	float sum = 0;
	if(x > 0 && x < xSize && y > 0 && y < ySize)
		switch (type) {
			case eChemicals::A:
				sum += pCurrentGrid[y * xSize + x].a * -1;
				sum += pCurrentGrid[y * xSize + (x - 1)].a * 0.2;
				sum += pCurrentGrid[y * xSize + (x + 1)].a * 0.2;
				sum += pCurrentGrid[(y + 1) * xSize + x].a * 0.2;
				sum += pCurrentGrid[(y - 1) * xSize + x].a * 0.2;	
				sum += pCurrentGrid[(y - 1) * xSize + (x - 1)].a * 0.05;
				sum += pCurrentGrid[(y - 1) * xSize + (x + 1)].a * 0.05;
				sum += pCurrentGrid[(y + 1) * xSize + (x - 1)].a * 0.05;
				sum += pCurrentGrid[(y + 1) * xSize + (x + 1)].a * 0.05;

				break;
			case eChemicals::B:
				sum += pCurrentGrid[y * xSize + x].b * -1;
				sum += pCurrentGrid[y * xSize + (x - 1)].b * 0.2;
				sum += pCurrentGrid[y * xSize + (x + 1)].b * 0.2;
				sum += pCurrentGrid[(y + 1) * xSize + x].b * 0.2;
				sum += pCurrentGrid[(y - 1) * xSize + x].b * 0.2;
				sum += pCurrentGrid[(y - 1) * xSize + (x - 1)].b * 0.05;
				sum += pCurrentGrid[(y - 1) * xSize + (x + 1)].b * 0.05;
				sum += pCurrentGrid[(y + 1) * xSize + (x - 1)].b * 0.05;
				sum += pCurrentGrid[(y + 1) * xSize + (x + 1)].b * 0.05;
		
				break;
		}	

	return sum;
}

__device__ float reactionA(float valueA, float valueB, int x, int y, Cell *pCurrentGrid, float killRate, float feedRate, float diffusionA, float dT, size_t xSize, size_t ySize) { 
	return valueA + ((diffusionA * laplacian(eChemicals::A, x, y, pCurrentGrid, xSize, ySize) * valueA) - (valueA * valueB * valueB) + (feedRate * (1 - valueA))) * dT;
}

__device__ float reactionB(float valueA, float valueB, int x, int y, Cell *pCurrentGrid, float killRate, float feedRate, float diffusionB ,float dT, size_t xSize, size_t ySize) { 
	return valueB + ((diffusionB * laplacian(eChemicals::B, x, y, pCurrentGrid, xSize, ySize) * valueB) + (valueA * valueB * valueB) - ((killRate + feedRate) * valueB)) * dT;
}

__global__ void setColor(sf::Uint8 *pPixelField, Cell* pCurrentGrid, size_t xSize) { 
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	Cell& currCell = pCurrentGrid[y * xSize + x];
	
	size_t pixelAddress = 4 * (y * xSize + x);

	pPixelField[pixelAddress + 0] = std::floor(currCell.a * 255);
	pPixelField[pixelAddress + 1] = std::floor((currCell.a + currCell.b) * 255);
	pPixelField[pixelAddress + 2] = std::floor(currCell.b * 255);
	pPixelField[pixelAddress + 3] = 255;
}

__global__ void ComputeState(Cell *pCurrentGrid, Cell *pNextGrid, size_t xSize, size_t ySize, float diffusionA, float diffusionB, float killRate, float feedRate, float dT) { 
	
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int cellAddress = y * xSize + x;

	Cell& prevState = pCurrentGrid[cellAddress];
	Cell& nextState = pNextGrid[cellAddress];

	nextState.a = reactionA(prevState.a, prevState.b, x, y, pCurrentGrid, killRate, feedRate, diffusionA, dT, xSize, ySize);
	nextState.b = reactionB(prevState.a, prevState.b, x, y, pCurrentGrid, killRate, feedRate, diffusionB, dT, xSize, ySize);
}

__global__ void setBValue(int xPos, int yPos, size_t xSize, Cell *pCurrentGrid) { 
	pCurrentGrid[yPos * xSize + xPos].b = 1;
}

void CudaSetSeed(int x, int y) {
	dim3 threadsPerBlock(cConfig.xThreads, cConfig.yThreads);
	dim3 numBlocks(xSize/threadsPerBlock.x, ySize/threadsPerBlock.y);

	setBValue<<<numBlocks, threadsPerBlock>>>(x, y, xSize, pCurrentGrid);
}

void CudaResetGrid() {
	hipMemset(pCurrentGrid, 0, xSize * ySize * sizeof(Cell));
	hipMemset(pNextGrid, 0, xSize * ySize * sizeof(Cell));
}

void CudaComputeField(sf::Uint8 *pResult) {

	dim3 threadsPerBlock(cConfig.xThreads, cConfig.yThreads);
	dim3 numBlocks(xSize/threadsPerBlock.x, ySize/threadsPerBlock.y);

	ComputeState<<<numBlocks, threadsPerBlock>>>(pCurrentGrid, pNextGrid, xSize, ySize, diffusionA, diffusionB, killRate, feedRate, dT);

	std::swap(pCurrentGrid, pNextGrid);

	setColor<<<numBlocks, threadsPerBlock>>>(pPixelField, pCurrentGrid, xSize);

	//hipDeviceSynchronize();
	hipMemcpy(pResult, pPixelField, ySize * xSize * sizeof(sf::Uint8) * 4, hipMemcpyDeviceToHost);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		std::cerr << hipGetErrorName(err) << std::endl;
}

